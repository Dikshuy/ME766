#include "hip/hip_runtime.h"
#include<bits/stdc++.h>

using namespace std;

#define BLOCK_SIZE 16

__global__ void matrix_multiplication(int *dev_a, int *dev_b, int *dev_c, int n){
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y*BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x*BLOCK_SIZE + threadIdx.x;
    int temp = 0;
    int idx;

    for (int i=0; i<gridDim.x; ++i){
        idx = row*n + i*BLOCK_SIZE + threadIdx.x;
        if (idx >= n*n){
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else{
            tile_a[threadIdx.y][threadIdx.x] = dev_a[idx];
        }

        idx = (i*BLOCK_SIZE + threadIdx.y)*n + col;
        if (idx >= n*n){
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }
        else{
            tile_b[threadIdx.y][threadIdx.x] = dev_b[idx];
        }
        __syncthreads();

        for (int j=0; j<BLOCK_SIZE; ++j){
            temp += tile_a[threadIdx.y][j]*tile_b[j][threadIdx.x];
        }
        __syncthreads();
    }
    if (row<n && col<n){
        dev_c[row*n+col] = temp;
    }
}

int main(int argc, char const *argv[]){
    int n;
    srand(1);
    int *a, *b, *c;
    n=10000;
    hipHostMalloc((void **) &a, sizeof(int)*n*n);
    hipHostMalloc((void **) &b, sizeof(int)*n*n);
    hipHostMalloc((void **) &c, sizeof(int)*n*n);

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            a[i * n + j] = rand() % n;
            b[i * n + j] = rand() % n;
        }
    }

    float time_taken;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);

    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void **) &dev_a, sizeof(int)*n*n);
    hipMalloc((void **) &dev_b, sizeof(int)*n*n);
    hipMalloc((void **) &dev_c, sizeof(int)*n*n);

    hipMemcpy(dev_a, a, sizeof(int)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(int)*n*n, hipMemcpyHostToDevice);

    unsigned int grid_rows = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
    unsigned int grid_cols = (n+BLOCK_SIZE-1)/BLOCK_SIZE;

    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    matrix_multiplication<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, n);

    hipMemcpy(c, dev_c, sizeof(int)*n*n, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_taken, start, stop);

    printf("Time elapsed in matrix multiplication on GPU: %f ms.\n",time_taken);

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}